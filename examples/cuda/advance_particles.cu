#include "hip/hip_runtime.h"

#include "particle.hpp"
#include <stdlib.h>
#include <stdio.h>

__global__ void advance_particles(float dt, particle* pArray, int nParticles)
{
    int idx = threadIdx.x + blockIdx.x*blockDim.x;
    if(idx < nParticles)
        pArray[idx].advance(dt);
}
