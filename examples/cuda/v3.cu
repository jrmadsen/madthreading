#include "hip/hip_runtime.h"

#include "v3.hpp"
#include "randomize.hpp"
#include <cmath>

v3::v3()
{
    ::randomize(x, y, z);
}

v3::v3(float xIn, float yIn, float zIn) : x(xIn), y(yIn), z(zIn)
{}

void v3::randomize()
{
    ::randomize(x, y, z);
}

__host__ __device__
void v3::normalize()
{
    float t = sqrt(x*x + y*y + z*z);
    x /= t;
    y /= t;
    z /= t;
}

__host__ __device__
void v3::scramble()
{
    float tx = 0.317f*(x + 1.0) + y + z * x * x + y + z;
    float ty = 0.619f*(y + 1.0) + y * y + x * y * z + y + x;
    float tz = 0.124f*(z + 1.0) + z * y + x * y * z + y + x;
    x = tx;
    y = ty;
    z = tz;
}

v3& v3::operator+=(const v3& rhs)
{
    x += rhs.x;
    y += rhs.y;
    z += rhs.z;
    return *this;
}

