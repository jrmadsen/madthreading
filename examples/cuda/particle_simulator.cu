
#include "particle_simulator.hpp"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include "particle.hpp"
#include "advance_particles.hpp"
#include <thread>
#include <string>
#include <sstream>

std::string get_context_info()
{
    std::stringstream ss;
    ss << "<CUDA Device=" << 0
       << ", Context=N/A"
       << ", Thread=" << std::this_thread::get_id()
       << ">";
    return ss.str();
}

v3_data simulate_particles(int32_t nitr, int64_t n, int16_t stream)
{
    printf("%s - %s() Launched...\n",
           get_context_info().c_str(), __FUNCTION__);

    hipError_t error;
    error = hipGetLastError();
    if (error != hipSuccess)
    {
        printf("0 %s\n", hipGetErrorString(error));
        exit(1);
    }

    particle* pArray = new particle[n];
    void* devPArray = NULL;
    hipMalloc(&devPArray, n*sizeof(particle));
    hipDeviceSynchronize();
    error = hipGetLastError();
    if (error != hipSuccess)
    {
        printf("1 %s\n", hipGetErrorString(error));
        exit(1);
    }

    hipMemcpy(devPArray, pArray, n*sizeof(particle), hipMemcpyHostToDevice);
    //hipDeviceSynchronize();
    error = hipGetLastError();
    if (error != hipSuccess)
    {
        printf("2 %s\n",hipGetErrorString(error));
        exit(1);
    }

    printf("\t%s - %s() running kernel...\n",
           get_context_info().c_str(), __FUNCTION__);
    for(int i=0; i < nitr; ++i)
    {
        float dt = (*rng::instance())(); // Random distance each step
        advance_particles<<< 1 +  n/64, 64, stream>>>(
                dt, (particle*) devPArray, n);
        error = hipGetLastError();
        if (error != hipSuccess)
        {
            printf("3 %s\n",hipGetErrorString(error));
            exit(1);
        }

        //hipDeviceSynchronize();
    }
    printf("\t%s - %s() finished kernel...\n",
           get_context_info().c_str(), __FUNCTION__);

    hipMemcpy(pArray, devPArray, n*sizeof(particle), hipMemcpyDeviceToHost);

    v3 total_distance(0,0,0);
    v3 temp;
    for(int i=0; i<n; i++)
    {
        temp = pArray[i].getTotalDistance();
        total_distance.x += temp.x;
        total_distance.y += temp.y;
        total_distance.z += temp.z;
    }

    printf("%s - %s() Finished...\n",
           get_context_info().c_str(), __FUNCTION__);

    v3_data _data;
    _data.num_entries = n;
    _data.total_distance = total_distance;

    return _data;
}
